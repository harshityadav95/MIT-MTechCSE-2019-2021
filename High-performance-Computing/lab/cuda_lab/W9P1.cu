#include "hip/hip_runtime.h"

#include<stdio.h>
#include<string.h>

__global__ void sort(char *s, int n, char *b)
{
	int id,i,c=0;
	id = threadIdx.x;
	for(i=0;i<n;i++)
	{
		if(id==i)
		continue;
		else if(s[i]<s[id])
		{
		c++;
		}
		b[c]=s[id];
	}
	
}

int main(void)
{
	char s[100],b[100];
	int n,i;
	printf("Enter the string\n");
	scanf("%s", s);

	n = strlen(s);
	char *d_s, *d_b;
	int size;

	size = sizeof(char);

	hipMalloc((void**)&d_s, n*size);
	hipMalloc((void**)&d_b, n*size);

	hipMemcpy(d_s,s,n*size, hipMemcpyHostToDevice);

	sort<<<1,n>>>(d_s,n,d_b);

	hipMemcpy(b,d_b,n*size, hipMemcpyDeviceToHost);

	for(i=0;i<n;i++)
	printf("%c",b[i]);

	hipFree(d_s);
	hipFree(d_b);

	return 0;
}