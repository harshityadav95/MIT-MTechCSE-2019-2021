#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>

__global__ void S_Sort(char* a,char* b,int n,int* pos)
{
	int i=0,k=0,strIn=0;
	int tid = threadIdx.x;
	if(a[tid] == ' ')
	{
		b[tid] = ' ';
	}
	else{
	for (i = 0; i < tid; i++)
	{
		if (a[i] == ' ')
		{
			k++;
			//printf("%d str %d\n",tid,strIn);
		}
	}
	char min;
	min = a[tid];
	strIn = pos[k];
	int nstrIn = pos[k + 1];
	if (nstrIn == 0)
	{
		nstrIn = n+1;
	}
	int inc=0;
	for (i = strIn; i < nstrIn-1;i++)
	{
		if (a[i] < min)
		{
			inc++;printf("min %c tid %d\n",min,tid);
		}
	}
	printf("b= %d ; a[tid]=%d\n",inc+strIn,tid);
	b[inc+strIn] = a[tid];
}
	
}

void MyWords(char* arr, int* n,int len)
{
	int i=0,k=0;
	n[k++] = 0;
	while (i != len)
	{
		if (arr[i++] == ' ')
		{
			n[k++] = i;
		}
	}
	n[k] = len;
}
int main(void)
{
	int N = 0, i, sz, strLth;
	int pos[20];
	char* d_a;char* d_c; int* d_pos;

	printf("Enter no of words in string:\n");
	scanf("%d", &N);
	char A[50],C[50];
	printf("Enter string:\n");
	scanf(" %[^\n]s", A);
	strLth = strlen(A);
	printf("Strlen %d", strLth);
	MyWords(A, pos,strLth);
	for (i = 0;i < N+1 ; i++)
	{
		printf(" pos %d =%d\t", i, pos[i]);
	}
	sz = sizeof(char) * strLth;
	hipMalloc((void**)&d_a, sz);
	hipMalloc((void**)&d_pos, N * sizeof(int));
	hipMalloc((void**)&d_c, sz);

	hipMemcpy(d_a, A, sz, hipMemcpyHostToDevice);
	hipMemcpy(d_pos, pos, N * sizeof(int), hipMemcpyHostToDevice);
	

	S_Sort << <1,strLth >> > (d_a,d_c,strLth,d_pos);

	hipMemcpy(C, d_c, sz, hipMemcpyDeviceToHost);
	for(i=0;i<strLth;i++)
	{
		printf("%c", C[i]);
	}
	

	hipFree(d_a);
	hipFree(d_pos);
	hipFree(d_c);
}