#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void Even(int* a,int n)
{
	int k;
	int tid = threadIdx.x;
	if(tid%2 == 0 && tid != n-1)
	{
		//printf("etid = %d\n",tid);
		if(a[tid] > a[tid+1]){//printf("even : %d\n",a[tid]);
		k = a[tid];
		a[tid] = a[tid+1];
		a[tid+1] = k;
		}
	}
}
__global__ void Odd(int* a,int n)
{
	int k;
	int tid = threadIdx.x;
	if(tid%2 != 0 && tid != n-1)
	{
	//printf("otid = %d\n",tid);
	if(a[tid] > a[tid+1]){
		//printf("odd : %d\n",a[tid]);
		k = a[tid];
		a[tid] = a[tid+1];
		a[tid+1] =  k;
	}
	}
}
int main(void)
{
	int N = 0, i, sz;
	int* d_a;

	printf("Enter Array size:\n");
	scanf("%d", &N);

	int A[100];
	printf("Enter string:\n");

	for (i = 0; i < N; i++)
	{
		A[i] = N-i;
		//scanf("%d", &A[i]);
		printf("%d ",A[i]);
	}
	printf("\n");

	sz = sizeof(int) * N;
	hipMalloc((void**)&d_a, sz);

	hipMemcpy(d_a, A, sz, hipMemcpyHostToDevice);
	int val = N/2;
	if(N%2 != 0)
	{
		val = val + 1;
	}

	for(i=0;i<val;i++)
	{
		Odd<<<1, N>>>(d_a,N);
		Even<<<1, N>>>(d_a,N);
	}
	hipMemcpy(A, d_a, sz, hipMemcpyDeviceToHost);
	for (i = 0; i < N; i++)
	{
		printf("%d ", A[i]);
	}
	printf("\n");
	hipFree(d_a);
}