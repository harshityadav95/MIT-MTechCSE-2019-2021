#include<stdio.h>
#include"hip/hip_runtime.h"

__global__ void add(int *a,int *b,int *c)
{
	*c=*a+*b;
}
int main()
{	
	int a,b,c;
	printf("\nValue of A:");
	scanf("%d",&a);
	printf("\nValue of b:");
	scanf("%d",&b);

	int *d_a,*d_b,*d_c;
	int size=sizeof(int);
	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_b,size);
	hipMalloc((void**)&d_c,size);
	hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);
	add<<<1,1>>>(d_a,d_b,d_c);
	hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);
	printf("The Sum of the Numbers is %d:",c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 1;




}	