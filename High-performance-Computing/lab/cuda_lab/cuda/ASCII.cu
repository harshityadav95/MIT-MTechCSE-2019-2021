#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
__global__ void ascii(char *a, int *b){
	int tid = threadIdx.x;
	b[tid] = int(a[tid]);
	printf("%d\t", b[tid]);
	printf("\n");
}
__global__ void reverse(int *b, int *c){
	int tid = threadIdx.x;
	c[tid]=0;
	while(b[tid] != 0)
    {
        c[tid] = c[tid]*10 + b[tid]%10;
        b[tid] /= 10;
    }
    printf("%d\t", c[tid]);
    printf("\n");
}
int main(void){
	int n, i, size, size1, b[100], *d_b, *d_c, c[100];
	char a[100], *d_a;
	printf("Enter the string to be toggled:\n");
	scanf("%s",a);
	n=strlen(a);
	//for(i=0; i<n; i++)
	//{
	//	b[i] = int(a[i]);
	//}
	size = sizeof(char);
	size1 = sizeof(int);
	hipMalloc((void **)&d_a,size*n);
	hipMalloc((void **)&d_b,size1*n);
	hipMalloc((void **)&d_c,size1*n);
	hipMemcpy(d_a,a,size*n,hipMemcpyHostToDevice);
	ascii <<<1,n>>> (d_a, d_b);
	hipMemcpy(b,d_b,size1*n,hipMemcpyDeviceToHost);
	for(i=0; i<n; i++)	
		{printf("%d\t",b[i]);}
	printf("\n");
	hipMemcpy(d_b,b,size1*n,hipMemcpyHostToDevice);
	reverse <<<1,n>>> (d_b, d_c);
	hipMemcpy(c,d_c,size1*n,hipMemcpyDeviceToHost);
	for(i=0; i<n; i++)
		{
			if(c[i]>26 && c[i]<126)
				{printf("%c\t",c[i]);}
			else
				{printf("%d\t",c[i]);}
		}
	printf("\n");
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}