#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
__global__ void toggle(char *a){
	int tid = threadIdx.x;
	if (a[tid] >= 'A' && a[tid] <= 'Z')
		{a[tid]+=32;}
	else if(a[tid] >= 'a' && a[tid] <= 'z')
		{a[tid] -=32;}
}
int main(void){
	int n,size;
	char a[100],*d_a;
	printf("Enter the string to be toggled:\n");
	scanf("%s",a);
	n=strlen(a);
	size = sizeof(char);
	hipMalloc((void **)&d_a,size*n);
	hipMemcpy(d_a,a,size*n,hipMemcpyHostToDevice);
	toggle <<<1,n>>> (d_a);
	hipMemcpy(a,d_a,size*n,hipMemcpyDeviceToHost);
		printf("%s",a);
	printf("\n");
	hipFree(d_a);
	return 0;
}
