#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a,int *b,int*c){
	int tid = threadIdx.x;
	c[tid] = a[tid] + b[tid];
}

int main(void){
	int n,a[1000],b[1000],c[1000],i,size,*d_a,*d_b,*d_c;
	printf("Enter no. of elements:\n");
	scanf("%d",&n);
	for(i=0;i<n;i++){
		a[i] = i;
		b[i] = i*2;
	}
	size = sizeof(int);
	hipMalloc((void **)&d_a,size*n);
	hipMalloc((void **)&d_b,size*n);
	hipMalloc((void **)&d_c,size*n);

	hipMemcpy(d_a,a,size*n,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size*n,hipMemcpyHostToDevice);

	add <<<1,n>>> (d_a,d_b,d_c);

	hipMemcpy(c,d_c,size*n,hipMemcpyDeviceToHost);
	for(i=0;i<n;i++)
		printf("%d\t",c[i]);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}