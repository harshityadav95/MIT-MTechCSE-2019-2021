#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>

__global__ void add(int *a,int *b){
	int tid = threadIdx.x;
	int y=0,z,i=1;
	z = a[tid];
	while(z!=0){
		y += (z % 8)*i;
		i = i*10;
		z = z/8;
	}
	b[tid] = y;
}

int main(void){
	int n,a[1000],b[1000],i,size,*d_a,*d_b;
	printf("Enter no. of elements:\n");
	scanf("%d",&n);
	for(i=0;i<n;i++){
		//scanf("%d",&a[i]);
		a[i] = rand()/100000;
		printf("%d\t",a[i]);
	}
	printf("\n");
	size = sizeof(int);
	hipMalloc((void **)&d_a,size*n);
	hipMalloc((void **)&d_b,size*n);

	hipMemcpy(d_a,a,size*n,hipMemcpyHostToDevice);

	add <<<1,n>>> (d_a,d_b);
	
	hipMemcpy(b,d_b,size*n,hipMemcpyDeviceToHost);
	for(i=0;i<n;i++)
		printf("%d\t",b[i]);
	printf("\n");
	hipFree(d_a);
	hipFree(d_b);
	return 0;
}