#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a,int *b){
	int tid = threadIdx.x;
	if(tid %2 ==0){
		if(a[tid+1]!=NULL){
			b[tid+1] = a[tid];
			b[tid] = a[tid+1];
		}
	}
}

int main(void){
	int n,i,size,*d_a,*d_b;
	int a[1000],b[1000];
	printf("Enter no. of elements:\n");
	scanf("%d",&n);
	for(i=0;i<n;i++){
		a[i] = i;
	}
	size = sizeof(int);
	hipMalloc((void **)&d_a,size*n);
	hipMalloc((void **)&d_b,size*n);

	hipMemcpy(d_a,a,size*n,hipMemcpyHostToDevice);

	add <<<1,n>>> (d_a,d_b);

	hipMemcpy(b,d_b,size*n,hipMemcpyDeviceToHost);
	for(i=0;i<n;i++)
		printf("%d\t",b[i]);
	printf("\n");
	hipFree(d_a);
	hipFree(d_b);
	return 0;
}