#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void matadd(int *a,int *b,int*c,int m,int n){
	int tid = threadIdx.x;
	//int sum,i,j;
	c[tid] = a[tid] + b[tid];
}

int main(void){
	int m,n,a[100],b[100],c[100],i,size,*d_a,*d_b,*d_c;
	printf("Enter M:\n");
	scanf("%d",&m);
	printf("Enter N:\n");
	scanf("%d",&n);
	printf("Enter matrix A:\n");
	for(i=0;i<m*n;i++){
		scanf("%d",&a[i]);
	}
	printf("Enter matrix B:\n");
	for(i=0;i<m*n;i++){
		scanf("%d",&b[i]);
	}
	size = sizeof(int);
	hipMalloc((void **)&d_a,size*n*m);
	hipMalloc((void **)&d_b,size*n*m);
	hipMalloc((void **)&d_c,size*n*m);

	hipMemcpy(d_a,a,size*n*m,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size*n*m,hipMemcpyHostToDevice);
	hipMemcpy(d_c,c,size*n*m,hipMemcpyHostToDevice);

	matadd <<<1,m*n>>> (d_a,d_b,d_c,m,n);

	hipMemcpy(c,d_c,size*n*m,hipMemcpyDeviceToHost);
	for(i=0;i<m*n;i++){
		printf("%d\t",c[i]);
	}
	printf("\n");
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}