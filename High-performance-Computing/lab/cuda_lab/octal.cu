#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void octal(int *a,int *b)
{
	int tid = threadIdx.x;
	long int rem[50],i=0,length=0,x=0;
	int num=a[tid];
	while(num>7)
 	{
 		rem[i]=num%8;
 		num=num/8;
 		i++;
 		length++;
 	}

	for(i=length-1;i>=0;i--)
	{
		x=x+(rem[i]*(10*i));
	}	
	b[tid]=x;

}
int main(void)
{
	int i,a[10000],b[10000],n;
	printf("Enter value of N:");
	scanf("%d",&n);
	printf("Enter array elements of array A\n");
	for(i=0;i<n;i++)
		scanf("%d",&a[i]);
	int *d_a,*d_b;
	int size = sizeof(int);
	hipMalloc((void **)&d_a,size*n);
	hipMalloc((void **)&d_b,size*n);
	hipMemcpy(d_a,a,size*n,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size*n,hipMemcpyHostToDevice);
	octal<<<1,n>>>(d_a,d_b);
	hipMemcpy(b,d_b,size*n,hipMemcpyDeviceToHost);
	printf("Octal array is :");
	for(i=0;i<n;i++)
	{
		printf("%d\t",b[i]);
	}
	hipFree(d_a);
	hipFree(d_b);

	return 0;
}
