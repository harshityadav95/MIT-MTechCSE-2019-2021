#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void star(char *a, int *n)
{   int i, j, t;
    int k = *n;
	int tid=threadIdx.x;
	j = k - tid - 1;
	t = (tid + 1) * 2 - 1;
	for(i = 0; i < t; i++){
	 a[tid * (k * 2 - 1) + j + i] = '*';
	}
}

int main(void)
{
	int i, N, j;
	scanf("%d", &N);
	
	int l = 2 * N - 1;
	char a[l][l];
	char *d_a;
	 int *d_b;
	int size=sizeof(char);
	int s = sizeof(int);
	hipMalloc((void **)&d_a,size*l*l);
	hipMalloc((void **)&d_b,s);
	hipMemcpy(d_a,&a,size*l*l,hipMemcpyHostToDevice);
	hipMemcpy(d_b,&N,s,hipMemcpyHostToDevice);
	star<<<1,N>>>(d_a,d_b);
	hipMemcpy(a,d_a,size*l*l,hipMemcpyDeviceToHost);
	for(i=0;i<l;i++)
	{
	for(j = 0; j < l; j++){
	if(a[i][j] == '*'){
	printf("%c", a[i][j]);
	}else{
	printf(" ");
	}
	}
	printf("\n");
	}
	return 0;
	}
