#include<stdio.h>
#include"hip/hip_runtime.h"

__global__ void add(int *a,int *b,int *c)
{
	int tid=threadIdx.x;
	c[tid]=a[tid]+b[tid];
}
int main()
{	

	int n,a[10],b[10],c[10];
	printf("\nValue of N:");
	scanf("%d",&n);
	printf("\n Enter the Values of array A:");
	for (int i = 0; i < n; ++i)
	{
		scanf("%d",&a[i]);
	}
	printf("\n Enter the Value of array B :");
	for (int i = 0; i < n; ++i)
	{
		scanf("%d",&b[i]);
	}
	int *d_a,*d_b,*d_c;
	int size=sizeof(int);
	hipMalloc((void**)&d_a,size*n);
	hipMalloc((void**)&d_b,size*n);
	hipMalloc((void**)&d_c,size*n);
	hipMemcpy(d_a,a,size*n,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size*n,hipMemcpyHostToDevice);
	add<<<1,n>>>(d_a,d_b,d_c);
	hipMemcpy(c,d_c,size*n,hipMemcpyDeviceToHost);
	printf("The Sum of the array is :");
	for (int i = 0; i < n; ++i)
	{
		printf("%d ",c[i]);
	}
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 1;




}	