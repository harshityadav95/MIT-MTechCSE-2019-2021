
#include <hip/hip_runtime.h>

__global__ void add(int *a,int *b,int *c)
{
	int tid = threadIdx.x;
	c[tid] = a[tid] + b[tid];
}
int main(void)
{
	int a[100],b[100],c[100],n;
	printf("Enter value of N:");
  n=3;
	printf("Enter array elements of array A\n");
  for(int i=0;i<n;i++)
  {
      a[i]=i;
  }
	printf("Enter array elements of array B\n");
	 for(int i=0;i<n;i++)
  {
      b[i]=i;
  }



int *d_a,*d_b,*d_c;
int size = sizeof(int);
hipMalloc((void **)&d_a,size*n);
hipMalloc((void **)&d_b,size*n);
hipMalloc((void **)&d_c,size*n);

hipMemcpy(d_a,a,size*n,hipMemcpyHostToDevice);
hipMemcpy(d_b,b,size*n,hipMemcpyHostToDevice);
add<<<1,n>>>(d_a,d_b,d_c);
hipMemcpy(c,d_c,size*n,hipMemcpyDeviceToHost);
printf("Sum array is :");



    	 for(int i=0;i<n;i++)
  {
           printf("%d ",c[i]);
  }

    

hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}