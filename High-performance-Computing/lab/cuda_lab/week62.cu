
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a,int *c)
{
	int tid = threadIdx.x;
    int n= a[tid] ;

  
    int temp=0,temp1=0;
    while(n!=0)
    {
        int t=n%8;
        temp=temp*10+t;
        n=n/8;
    }
    while(temp!=0)
    {
        int t=temp%10;
        temp1=temp1*10+t;
        temp=temp/10;
    }
    c[tid]=temp1;
    
    
}
int main(void)
{
	int n,a[20],c[20];
	printf("Enter value of N:");
  n=5;
	printf("Enter array elements of array A\n");
  for(int i=0;i<n;i++)
  {
      a[i]=i;
  }
	
int *d_a,*d_c;
int size = sizeof(int);
hipMalloc((void **)&d_a,size*n);
hipMalloc((void **)&d_c,size*n);
hipMemcpy(d_a,a,size*n,hipMemcpyHostToDevice);
add<<<1,n>>>(d_a,d_c);
hipMemcpy(c,d_c,size*n,hipMemcpyDeviceToHost);
printf("Octal array is :");
for(int i=0;i<n;i++)
{
           printf("%d ",c[i]);
}

    

hipFree(d_a);
hipFree(d_c);
return 0;
}