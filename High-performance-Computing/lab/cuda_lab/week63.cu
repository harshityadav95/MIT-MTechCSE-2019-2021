
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a,int *b)
{
  int tid = threadIdx.x;
  int n=a[tid];
  if(tid+2<*b && tid<(*b)/2)
    {
        a[tid]=a[tid+2];
        a[tid+2]=n;
    }
 
    
}
int main(void)
{
  int n,a[20],c[20];
  printf("Enter value of N:");
  n=5;
  printf("Enter array elements of array A\n");
  for(int i=0;i<n;i++)
  {
      a[i]=i;
  }
  
int *d_a,*d_b,*d;
int size = sizeof(int);
hipMalloc((void **)&d_a,size*n);
hipMalloc((void **)&d_b,size);
    
hipMemcpy(d_a,a,size*n,hipMemcpyHostToDevice);
hipMemcpy(d_b,&n,size,hipMemcpyHostToDevice);
add<<<1,n>>>(d_a,d_b);
hipMemcpy(c,d_a,size*n,hipMemcpyDeviceToHost);
printf("array is :");
for(int i=0;i<n;i++)
{
           printf("%d ",c[i]);
}

    

hipFree(d_a);
hipFree(d_b);
return 0;
}