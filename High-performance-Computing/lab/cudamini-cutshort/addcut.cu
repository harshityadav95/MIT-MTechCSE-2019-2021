
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<malloc.h>
#include<time.h>
#define arm 32
__device__ int globalArray[32];
__global__ void add(int *a,int *c)
{
	
    int tid = threadIdx.x;
    int temp=a[tid];
    int count=0;
    while(temp!=0)
    {
        count++;
        temp=temp/2;
    }
    atomicAdd(&globalArray[count], 1);
    c[count]=globalArray[count];

}
int main(void)
{
    int arr[20];
    int count=20;
    int bitband[arm]={0};
    for (int i = 0; i < 20; ++i)
    {
        arr[i]=i;
    }

int *d_a,*d_c;
int size = sizeof(int);
hipMalloc((void **)&d_a,size*count);

hipMalloc((void **)&d_c,size*arm);

hipMemcpy(d_a,arr,size*count,hipMemcpyHostToDevice);


add<<<1,count>>>(d_a,d_c);
hipMemcpy(bitband,d_c,size*arm,hipMemcpyDeviceToHost);
   for (int i = 0; i < 20; ++i)
   {
       printf("%d\n",bitband[i]);
   }

hipFree(d_a);
hipFree(d_c);
return 0;
}