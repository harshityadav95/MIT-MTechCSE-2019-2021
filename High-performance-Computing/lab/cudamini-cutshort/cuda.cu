#include "hip/hip_runtime.h"

#include <stdio.h>
#include<stdlib.h>
#include<stdio.h>
#include<malloc.h>
#include<time.h>
#define arm 32
__device__ int globalArray[32];
__global__ void add(int *a,int *c)
{
	
    int tid = threadIdx.x;
    int temp=a[tid];
    int count=0;
    while(temp!=0)
    {
        count++;
        temp=temp/2;
    }
    atomicAdd(&globalArray[count], 1);
    c[count]=globalArray[count];

}
int bitcount(int num);
void Sort(int arr[], int n) ;
void swap(int* a, int* b) 
{ 
    int t = *a; 
    *a = *b; 
    *b = t; 
} 
int partition (int arr[], int low, int high) 
{ 
    int pivot = arr[high];    // pivot 
    int i = (low - 1);  // Index of smaller element 
  
    for (int j = low; j <= high- 1; j++) 
    { 
        // If current element is smaller than the pivot 
        if (arr[j] < pivot) 
        { 
            i++;    // increment index of smaller element 
            swap(&arr[i], &arr[j]); 
        } 
    } 
    swap(&arr[i + 1], &arr[high]); 
    return (i + 1); 
} 
void quicksort(int arr[], int low, int high) 
{ 
    if (low < high) 
    { 
        /* pi is partitioning index, arr[p] is now 
           at right place */
        int pi = partition(arr, low, high); 
  
        // Separately sort elements before 
        // partition and after partition 
        quicksort(arr, low, pi - 1); 
        quicksort(arr, pi + 1, high); 
    } 
} 

int main(void)
{
	int arr[50000]={0};
	int bitband[arm]={0};
	int pos;
	int resultant[50000]={0};
	 int bitmap[arm]={0};

	int n=0;
	int count=0;
       ///a[20]={1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20};
        /*for ( int i = 0; i < 20; ++i)
        {
            arr[i]=i+1;
            count++;
        }*/
    	FILE *fptr;
    		int num;
    	fptr = fopen("integers", "r");
		printf("\ncounting the File:\n");
		while ( (num = getw(fptr)) != EOF ) 
		{
			
    		arr[count]=num;
    		count++;

  		}
  		fclose(fptr);
  		n=count;
  		/*for (int i = 0; i < count; ++i)
  		{
  			printf("%d\n",arr[i] );
  		}*/


int *d_a,*d_c;
int size = sizeof(int);
hipMalloc((void **)&d_a,size*n);

hipMalloc((void **)&d_c,size*arm);

hipMemcpy(d_a,arr,size*n,hipMemcpyHostToDevice);
clock_t start,stop;
start=clock();
add<<<1,n>>>(d_a,d_c);
hipMemcpy(bitband,d_c,size*arm,hipMemcpyDeviceToHost);
 for (int i = 0; i < 32; ++i)
   {
       printf("%d\n",bitband[i]);
   }

   for(int i=1;i<=arm;i++)
        {
            bitband[i] += bitband[i-1];
        }
          for(int i=0;i<count;i++)
        {
            pos=bitcount(arr[i]);
            resultant[ ((pos==0)?0:bitband[pos-1]) + bitmap[pos] ] = arr[i];
            bitmap[pos] += 1;
        }
        
     

        for(int i=1;i<=arm;i++)
        {
            quicksort(resultant,bitband[i-1],bitband[i]-1);
        }
        stop=clock(); 
        printf("\nSorting Completed in time");
        
      	printf("CLOCKS PER SECOND = %ld\n",CLOCKS_PER_SEC);
        printf("START CLOCK = %ld \nSTOP CLOCK = %ld \n",start,stop);
        printf("TIME TAKEN = %f\n",(float)(stop-start)/CLOCKS_PER_SEC);

hipFree(d_a);
hipFree(d_c);
return 0;
}
int bitcount(int num)
{
    unsigned int count1=0,count2=0, tmp,value=4278190080U,result;
    // checking the byte no. of register
    if(num==0)
        return 0;
    while(!(num & value))
    {
       value = value >> 8;
       count1++;
    }

    value = 2147483648U;
    tmp=count1;

    // moving the first bit to proper byte
    while(tmp--)
    {
        value = value >> 8;
    }

    // moving the bit in a particular byte
    while(!(num & value))
    {
        value = value >> 1;
        count2++;
    }
    result = 32 - (count1 * 8 + count2);
    return result;
}
void Sort(int arr[], int n)  
{  
    int i, j;  
    for (i = 0; i < n-1; i++)      
      
    // Last i elements are already in place  
    for (j = 0; j < n-i-1; j++)  
        if (arr[j] > arr[j+1])  
            swap(&arr[j], &arr[j+1]);  
}  

