#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a,int *b,int *c)
{
	int tid = threadIdx.x;
	c[tid] = a[tid] + b[tid];
}
int main(void)
{
	int i,a[10000],b[10000],c[10000],n;
	printf("Enter value of N:");
	scanf("%d",&n);
	printf("Enter array elements of array A\n");
	for(i=0;i<n;i++)
		scanf("%d",&a[i]);

	printf("Enter array elements of array B\n");
	for(i=0;i<n;i++)
		scanf("%d",&b[i]);	




int *d_a,*d_b,*d_c;
int size = sizeof(int);
hipMalloc((void **)&d_a,size*n);
hipMalloc((void **)&d_b,size*n);
hipMalloc((void **)&d_c,size*n);

hipMemcpy(d_a,a,size*n,hipMemcpyHostToDevice);
hipMemcpy(d_b,b,size*n,hipMemcpyHostToDevice);
add<<<1,n>>>(d_a,d_b,d_c);
hipMemcpy(c,d_c,size*n,hipMemcpyDeviceToHost);
printf("Sum array is :");
for(i=0;i<n;i++)
	printf("%d\t",c[i]);

hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}