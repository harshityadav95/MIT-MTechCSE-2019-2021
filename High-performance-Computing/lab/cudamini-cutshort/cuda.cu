#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<stdlib.h>
#include<stdio.h>
#include<malloc.h>
#include<time.h>
#define arm 32
__device__ int globalArray[32];
__global__ void add(int *a,int *c)
{
	
    int tid = threadIdx.x;
    int temp=a[tid];
    int count=0;
    while(temp!=0)
    {
        count++;
        temp=temp/2;
    }
    atomicAdd(&globalArray[count], 1);
    c[count]=globalArray[count];
    
	  

}
int bitcount(int num);
void swap(int* a, int* b) 
{ 
    int t = *a; 
    *a = *b; 
    *b = t; 
} 
int partition (int arr[], int low, int high) 
{ 
    int pivot = arr[high];    // pivot 
    int i = (low - 1);  // Index of smaller element 
  
    for (int j = low; j <= high- 1; j++) 
    { 
        // If current element is smaller than the pivot 
        if (arr[j] < pivot) 
        { 
            i++;    // increment index of smaller element 
            swap(&arr[i], &arr[j]); 
        } 
    } 
    swap(&arr[i + 1], &arr[high]); 
    return (i + 1); 
} 
void quicksort(int arr[], int low, int high) 
{ 
    if (low < high) 
    { 
        /* pi is partitioning index, arr[p] is now 
           at right place */
        int pi = partition(arr, low, high); 
  
        // Separately sort elements before 
        // partition and after partition 
        quicksort(arr, low, pi - 1); 
        quicksort(arr, pi + 1, high); 
    } 
} 

int main(void)
{
	int i,arr[50000]={0},bitband[arm]={0},n;
	int pos;
	 int resultant[50000]={0};
	 int bitmap[arm]={0};
	int num;
	int count=0;
       ///a[20]={1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20};
        /*for ( i = 0; i < count; ++i)
        {
            a[i]=i+1;
        }*/
    	FILE *fptr;
    	fptr = fopen("integers", "r");
		printf("\ncounting the File:\n");
		while ( (num = getw(fptr)) != EOF ) 
		{
    		arr[count]=num;
    		count++;

  		}
  		fclose(fptr);
  		n=count;


int *d_a,*d_b,*d_c;
int size = sizeof(int);
hipMalloc((void **)&d_a,size*n);

hipMalloc((void **)&d_c,size*arm);

hipMemcpy(d_a,a,size*n,hipMemcpyHostToDevice);
clock_t start,stop;
start=clock();
add<<<1,n>>>(d_a,d_c);
hipMemcpy(bitband,d_c,size*arm,hipMemcpyDeviceToHost);
   for(int i=1;i<=arm;i++)
        {
            bitband[i] += bitband[i-1];
        }
          for(int i=0;i<count;i++)
        {
            pos=bitcount(arr[i]);
            resultant[ ((pos==0)?0:bitband[pos-1]) + bitmap[pos] ] = arr[i];
            bitmap[pos] += 1;
        }

for(int i=0;i<count;i++)
        {
            pos=bitcount(arr[i]);
            resultant[ ((pos==0)?0:bitband[pos-1]) + bitmap[pos] ] = arr[i];
            bitmap[pos] += 1;
        }

        for(int i=1;i<=arm;i++)
        {
            quicksort(resultant,bitband[i-1],bitband[i]-1);
        }
        stop=clock();
     
        printf("CLOCKS PER SECOND = %ld\n",CLOCKS_PER_SEC);
        printf("START CLOCK = %ld \nSTOP CLOCK = %ld \n",start,stop);
        printf("TIME TAKEN = %f\n",(float)(stop-start)/CLOCKS_PER_SEC);


hipFree(d_a);
hipFree(d_c);
return 0;
}